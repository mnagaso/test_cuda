#include "hip/hip_runtime.h"

#include<iostream>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>
#include<vector>
#include "hipblas.h"
#include "Eigen/Core"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define cuda_error_check(x) do { if((x) != 0) { printf("cuda error in %s at line: %d\n", __FILE__, __LINE__); exit(1);} } while(false)

const int nbatch=100;
const int nrows = 3;
const int ncols = 3;
const int ltau  = 3;

int main(){
   // init n eigen matrices
    Eigen::MatrixXd a_mat(nrows, ncols);
    //a_mat << 12, -51, 4, \
    //        6, 167, -68, \
    //        -4, 24, -41;
    a_mat << 2.5,1.1,0.3, \
             2.2,1.9,0.4, \
             1.8,0.1,0.3;


    std::vector<Eigen::MatrixXd> e_mats;
    for (int i = 0; i < nbatch; i++){
        e_mats.push_back(a_mat);
    }
 
    // cublas init
    hipblasHandle_t handle;
    cuda_error_check(hipblasCreate(&handle));

    // array on host
    // use Eigen::Matrix::data() method to access the pointer array inside of Eigen obj
    // arrays on device
    double*  devPtrMat[nbatch];  
    double*  devPtrMat_TAU[nbatch];
    double** d_devPtrMat;
    double** d_devPtrMat_TAU;
    int info;


    for (int i = 0; i < nbatch; i++){
      cuda_error_check(hipMalloc((void**)&devPtrMat[i], nrows*ncols*sizeof(double)));
      cuda_error_check(hipMalloc((void**)&devPtrMat_TAU[i], ltau*sizeof(double)));
    }

    cuda_error_check(hipMalloc((void**)&d_devPtrMat,     sizeof(double*)*nbatch));
    cuda_error_check(hipMalloc((void**)&d_devPtrMat_TAU, sizeof(double*)*nbatch));
    
    // download memory
    for (int i = 0; i < nbatch; i++){
        double* mtmp = e_mats[i].data();
        cuda_error_check(hipblasSetMatrix (nrows, ncols, sizeof(double), mtmp, nrows, devPtrMat[i], nrows));
    }

    cuda_error_check(hipMemcpy(d_devPtrMat,     devPtrMat,     sizeof(devPtrMat),     hipMemcpyHostToDevice));
    cuda_error_check(hipMemcpy(d_devPtrMat_TAU, devPtrMat_TAU, sizeof(devPtrMat_TAU), hipMemcpyHostToDevice));

    // do qr
    cuda_error_check(hipDeviceSynchronize());
    cuda_error_check(hipblasDgeqrfBatched(handle, nrows, ncols, d_devPtrMat, ltau, d_devPtrMat_TAU, &info, nbatch));
    cuda_error_check(hipDeviceSynchronize());

    // upload memory
    double* tmp;
    double* tmp2;
    tmp =(double*)malloc(nrows*ncols*sizeof(double));
    tmp2=(double*)malloc(ltau*sizeof(double));


    for (int i = 0; i < nbatch; i++){
        cuda_error_check(hipMemcpy(tmp, devPtrMat[i], nrows*ncols*sizeof(double), hipMemcpyDeviceToHost));
        Eigen::Map<Eigen::MatrixXd> tmpm(tmp,nrows,ncols);            
        e_mats[i] = tmpm;

        cuda_error_check(hipMemcpy(tmp2,devPtrMat_TAU[i],ltau*sizeof(double), hipMemcpyDeviceToHost));
    }
    for (int i = 0; i < nbatch; i++){
        if (i == 0){
            std::cout << "i- " << i << std::endl;
            std::cout << e_mats[i] << std::endl;
            std::cout << "tau: ";
            for (int j = 0; j < ltau; j++) std::cout << tmp2[j] << "   ";
        }
    }

    free(tmp);
    free(tmp2);

    if (d_devPtrMat)     hipFree(d_devPtrMat);
    if (d_devPtrMat_TAU) hipFree(d_devPtrMat_TAU);
    if (handle)          hipblasDestroy(handle);
        

    return 0;
}
